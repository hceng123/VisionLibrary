#include "hip/hip_runtime.h"
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""
#include "CudaFunc.h"

__global__
void kernel_merge_height_intersect(float* matOne, float *matTwo, float *d_result, int rows, int cols, float fDiffThreshold) {
    int start = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    int *arrayIndexs = (int *)malloc(cols / 2 * sizeof(int));
    //assert(arrayIndexs != NULL);

    float *arrayTargetH = (float *)malloc(cols / 2 * sizeof(float));
    //assert(arrayTargetH != NULL);

    float *matTreRow = (float *)malloc(cols * sizeof(float));
    //assert(matTreRow != NULL);

    //float *matForRow = (float *)malloc(cols * sizeof(float));
    //assert(matForRow != NULL);

    char *matNanMaskRow = (char *)malloc(cols);
    //assert(matNanMaskRow != NULL);

    memset(matNanMaskRow, 0, cols);

    for (int row = start; row < rows; row += stride) {
        int offset = row * cols;
        float *matOneRow = matOne + offset;
        float *matTwoRow = matTwo + offset;
        float *matForRow = d_result + offset;

        for (int i = 0; i < cols; ++i) {
            matTreRow[i] = (matOneRow[i] + matTwoRow[i]) / 2;
            matForRow[i] = matTreRow[i];
            float absDiff = fabs(matOneRow[i] - matTwoRow[i]);
            if (isnan(absDiff) || absDiff > fDiffThreshold)
                matNanMaskRow[i] = 1;
        }

        //memcpy(matForRow, matTreRow, cols * sizeof(float));

        int count = 0;
        if (matNanMaskRow[0] > 0)
            arrayIndexs[count++] = 0;

        for (int col = 0; col < cols - 1; ++col) {
            if (fabsf(matNanMaskRow[col + 1] - matNanMaskRow[col]) > 0)
                arrayIndexs[count++] = col;
        }

        if (matNanMaskRow[cols - 1] > 0)
            arrayIndexs[count++] = cols - 1;

        for (int i = 0; i < count / 2; ++i) {
            int startIndex = arrayIndexs[i * 2];
            int endIndex = arrayIndexs[i * 2 + 1];

            if (0 == startIndex && endIndex < cols - 1) {
                for (int i = 0; i < endIndex - startIndex; ++i)
                    arrayTargetH[i] = matForRow[endIndex + 1];
            }
            else if (cols - 1 == endIndex && startIndex >= 0) {
                for (int i = 0; i < endIndex - startIndex; ++i)
                    arrayTargetH[i] = matForRow[startIndex];
            }
            else if (startIndex >= 0 && endIndex < cols - 1) {
                float fInterval = (matForRow[endIndex + 1] - matForRow[startIndex]) / (endIndex - startIndex + 1);
                for (int i = 0; i < endIndex - startIndex; ++i) {
                    arrayTargetH[i] = matForRow[startIndex] + fInterval * i;
                }
            }

            float fAbsDiffSumOne = 0.f, fAbsDiffSumTwo = 0.f, fAbsDiffSumTre = 0.f;
            for (int index = startIndex, k = 0; index < endIndex; ++index, ++k) {
                fAbsDiffSumOne += fabsf(matOneRow[index] - arrayTargetH[k]);
                fAbsDiffSumTwo += fabsf(matTwoRow[index] - arrayTargetH[k]);
                fAbsDiffSumTre += fabsf(matTreRow[index] - arrayTargetH[k]);
            }

            float *matChoose;
            if (fAbsDiffSumOne <= fAbsDiffSumTwo && fAbsDiffSumOne <= fAbsDiffSumTre)
                matChoose = matOneRow;
            else if (fAbsDiffSumTwo <= fAbsDiffSumOne && fAbsDiffSumTwo <= fAbsDiffSumTre)
                matChoose = matTwoRow;
            else
                matChoose = matTreRow;

            for (int col = startIndex; col <= endIndex; ++col) {
                matForRow[col] = matChoose[col];
            }
        }

        //memcpy(matOneRow, matForRow, cols * sizeof(float));
    }

    free(arrayTargetH);
    free(arrayIndexs);
    free(matTreRow);
    //free(matForRow);
    free(matNanMaskRow);
}

void run_kernel_merge_height_intersect(
    uint32_t gridSize,
    uint32_t blockSize,
    float* matOne,
    float *matTwo,
    float *d_result,
    int rows,
    int cols,
    float fDiffThreshold) {
    kernel_merge_height_intersect<<<gridSize, blockSize>>>(matOne, matTwo, d_result, rows, cols, fDiffThreshold);
}

__global__
void kernel_select_cmp_point(
    float* dMap,
    float* dPhase,
    uint8_t* matResult,
    const int ROWS,
    const int COLS,
    const int span) {
    int start = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    int *arrayIdx1 = (int *)malloc(COLS / 4 * sizeof(int));
    int *arrayIdx2 = (int *)malloc(COLS / 4 * sizeof(int));

    for (int row = start; row < ROWS; row += stride) {
        int offsetOfInput = row * COLS;
        int offsetOfResult = row * COLS;

        float* dMapRow = dMap + offsetOfInput;
        float* dPhaseRow = dPhase + offsetOfInput;
        uint8_t* matResultRow = matResult + offsetOfResult;

        int countOfIdx1 = 0, countOfIdx2 = 0;
        for (int i = 0; i < COLS - 1; ++ i) {
            const auto& value = dMapRow[i];
            if (value == 1.f)
                arrayIdx1[countOfIdx1++] = i;
            else if (value == -1.f)
                arrayIdx2[countOfIdx2++] = i;
        }

        if (countOfIdx1 > 0 && countOfIdx2 > 0) {
            // If start of index1 larger than index 2, then remove beginning element of index2
            if (arrayIdx1[0] > arrayIdx2[0]) {
                countOfIdx2--;
                for (int i = 0; i < countOfIdx2; ++i)
                    arrayIdx2[i] = arrayIdx2[i + 1];
            }

            if (countOfIdx1 > countOfIdx2) {
                countOfIdx1--;
            }
        }

        for (int i = 0; i < countOfIdx1 && i < countOfIdx2; ++i) {
            if (arrayIdx2[i] - arrayIdx1[i] < span) {
                if (fabs(dPhaseRow[arrayIdx1[i]]) > 1.f && fabs(dPhaseRow[arrayIdx2[i]]) > 1.f) {
                    for (int k = arrayIdx1[i]; k <= arrayIdx2[i]; ++k)
                        matResultRow[k] = 255;
                }
            }
        }
    }

    free(arrayIdx1);
    free(arrayIdx2);
}

void test_kernel_select_cmp_point(
    float* dMap,
    float* dPhase,
    uint8_t* matResult,
    const int ROWS,
    const int COLS,
    const int span) {
    int start = 0;
    int stride = 1;

    int *arrayIdx1 = (int *)malloc(COLS / 4 * sizeof(int));
    int *arrayIdx2 = (int *)malloc(COLS / 4 * sizeof(int));

    for (int row = start; row < ROWS; row += stride) {
        int offsetOfInput = row * COLS;
        int offsetOfResult = row * COLS;

        float* dMapRow = dMap + offsetOfInput;
        float* dPhaseRow = dPhase + offsetOfInput;
        uint8_t* matResultRow = matResult + offsetOfResult;

        int countOfIdx1 = 0, countOfIdx2 = 0;
        for (int i = 0; i < COLS - 1; ++ i) {
            const auto& value = dMapRow[i];
            if (value == 1.f)
                arrayIdx1[countOfIdx1++] = i;
            else if (value == -1.f)
                arrayIdx2[countOfIdx2++] = i;
        }

        if (countOfIdx1 > 0 && countOfIdx2 > 0) {
            // If start of index1 larger than index 2, then remove beginning element of index2
            if (arrayIdx1[0] > arrayIdx2[0]) {
                countOfIdx2--;
                for (int i = 0; i < countOfIdx2; ++i)
                    arrayIdx2[i] = arrayIdx2[i + 1];
            }

            if (countOfIdx1 > countOfIdx2) {
                countOfIdx1--;
            }
        }

        for (int i = 0; i < countOfIdx1 && i < countOfIdx2; ++i) {
            if (arrayIdx2[i] - arrayIdx1[i] < span) {
                if (fabs(dPhaseRow[arrayIdx1[i]]) > 1.f && fabs(dPhaseRow[arrayIdx2[i]]) > 1.f) {
                    for (int k = arrayIdx1[i]; k <= arrayIdx2[i]; ++k)
                        matResultRow[k] = 255;
                }
            }
        }
    }

    free(arrayIdx1);
    free(arrayIdx2);
}

void run_kernel_select_cmp_point(
    uint32_t gridSize,
    uint32_t blockSize,
    float* dMap,
    float* dPhase,
    uint8_t* matResult,
    const int ROWS,
    const int COLS,
    const int span) {
    kernel_select_cmp_point<<<gridSize, blockSize>>>(dMap, dPhase, matResult, ROWS, COLS, span);
    //test_kernel_select_cmp_point(dMap, dPhase, matResult, ROWS, COLS, span);
}
