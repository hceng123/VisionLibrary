#include "hip/hip_runtime.h"
#include <stdio.h>

#include ""
#include "CudaFunc.h"

template<typename T>
__device__ __host__
void sort2Value(T& a, T& b) {
    T t = a;
    a = min(a, b);
    b = max(b, t);
}

template<typename T>
__device__ __host__
T median9(T *p) {
    sort2Value(p[1], p[2]); sort2Value(p[4], p[5]); sort2Value(p[7], p[8]); sort2Value(p[0], p[1]);
    sort2Value(p[3], p[4]); sort2Value(p[6], p[7]); sort2Value(p[1], p[2]); sort2Value(p[4], p[5]);
    sort2Value(p[7], p[8]); sort2Value(p[0], p[3]); sort2Value(p[5], p[8]); sort2Value(p[4], p[7]);
    sort2Value(p[3], p[6]); sort2Value(p[1], p[4]); sort2Value(p[2], p[5]); sort2Value(p[4], p[7]);
    sort2Value(p[4], p[2]); sort2Value(p[6], p[4]); sort2Value(p[4], p[2]);
    return p[4];
}

template<typename T>
__device__ __host__
T median25(T *p) {
    sort2Value(p[1], p[2]); sort2Value(p[0], p[1]); sort2Value(p[1], p[2]); sort2Value(p[4], p[5]); sort2Value(p[3], p[4]);
    sort2Value(p[4], p[5]); sort2Value(p[0], p[3]); sort2Value(p[2], p[5]); sort2Value(p[2], p[3]); sort2Value(p[1], p[4]);
    sort2Value(p[1], p[2]); sort2Value(p[3], p[4]); sort2Value(p[7], p[8]); sort2Value(p[6], p[7]); sort2Value(p[7], p[8]);
    sort2Value(p[10], p[11]); sort2Value(p[9], p[10]); sort2Value(p[10], p[11]); sort2Value(p[6], p[9]); sort2Value(p[8], p[11]);
    sort2Value(p[8], p[9]); sort2Value(p[7], p[10]); sort2Value(p[7], p[8]); sort2Value(p[9], p[10]); sort2Value(p[0], p[6]);
    sort2Value(p[4], p[10]); sort2Value(p[4], p[6]); sort2Value(p[2], p[8]); sort2Value(p[2], p[4]); sort2Value(p[6], p[8]);
    sort2Value(p[1], p[7]); sort2Value(p[5], p[11]); sort2Value(p[5], p[7]); sort2Value(p[3], p[9]); sort2Value(p[3], p[5]);
    sort2Value(p[7], p[9]); sort2Value(p[1], p[2]); sort2Value(p[3], p[4]); sort2Value(p[5], p[6]); sort2Value(p[7], p[8]);
    sort2Value(p[9], p[10]); sort2Value(p[13], p[14]); sort2Value(p[12], p[13]); sort2Value(p[13], p[14]); sort2Value(p[16], p[17]);
    sort2Value(p[15], p[16]); sort2Value(p[16], p[17]); sort2Value(p[12], p[15]); sort2Value(p[14], p[17]); sort2Value(p[14], p[15]);
    sort2Value(p[13], p[16]); sort2Value(p[13], p[14]); sort2Value(p[15], p[16]); sort2Value(p[19], p[20]); sort2Value(p[18], p[19]);
    sort2Value(p[19], p[20]); sort2Value(p[21], p[22]); sort2Value(p[23], p[24]); sort2Value(p[21], p[23]); sort2Value(p[22], p[24]);
    sort2Value(p[22], p[23]); sort2Value(p[18], p[21]); sort2Value(p[20], p[23]); sort2Value(p[20], p[21]); sort2Value(p[19], p[22]);
    sort2Value(p[22], p[24]); sort2Value(p[19], p[20]); sort2Value(p[21], p[22]); sort2Value(p[23], p[24]); sort2Value(p[12], p[18]);
    sort2Value(p[16], p[22]); sort2Value(p[16], p[18]); sort2Value(p[14], p[20]); sort2Value(p[20], p[24]); sort2Value(p[14], p[16]);
    sort2Value(p[18], p[20]); sort2Value(p[22], p[24]); sort2Value(p[13], p[19]); sort2Value(p[17], p[23]); sort2Value(p[17], p[19]);
    sort2Value(p[15], p[21]); sort2Value(p[15], p[17]); sort2Value(p[19], p[21]); sort2Value(p[13], p[14]); sort2Value(p[15], p[16]);
    sort2Value(p[17], p[18]); sort2Value(p[19], p[20]); sort2Value(p[21], p[22]); sort2Value(p[23], p[24]); sort2Value(p[0], p[12]);
    sort2Value(p[8], p[20]); sort2Value(p[8], p[12]); sort2Value(p[4], p[16]); sort2Value(p[16], p[24]); sort2Value(p[12], p[16]);
    sort2Value(p[2], p[14]); sort2Value(p[10], p[22]); sort2Value(p[10], p[14]); sort2Value(p[6], p[18]); sort2Value(p[6], p[10]);
    sort2Value(p[10], p[12]); sort2Value(p[1], p[13]); sort2Value(p[9], p[21]); sort2Value(p[9], p[13]); sort2Value(p[5], p[17]);
    sort2Value(p[13], p[17]); sort2Value(p[3], p[15]); sort2Value(p[11], p[23]); sort2Value(p[11], p[15]); sort2Value(p[7], p[19]);
    sort2Value(p[7], p[11]); sort2Value(p[11], p[13]); sort2Value(p[11], p[12]);
    return p[12];
}

template<typename T>
__global__
void kernel_median_filter(
    const T* src,
    T* dst,
    const int ROWS,
    const int COLS,
    const int step,
    const int winSize) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    if (r >= ROWS || c >= COLS)
        return;
    if (3 == winSize) {
        const T* row0 = src + max(r - 1, 0) * step;
        const T* row1 = src + r * step;
        const T* row2 = src + min(r + 1, ROWS - 1) * step;

        int j0 = c >= 1 ? c - 1 : c;
        int j2 = c < COLS - 1 ? c + 1 : c;

        T p[9];
        p[0] = row0[j0], p[1] = row0[c], p[2] = row0[j2];
        p[3] = row1[j0], p[4] = row1[c], p[5] = row1[j2];
        p[6] = row2[j0], p[7] = row2[c], p[8] = row2[j2];

        dst[r * step + c] = median9(p);
    }
    else if (5 == winSize) {
        const T* row[5];
        row[0] = src + max(r - 2, 0) * step;
        row[1] = src + max(r - 1, 0) * step;
        row[2] = src + r * step;
        row[3] = src + min(r + 1, ROWS - 1) * step;
        row[4] = src + min(r + 2, ROWS - 1) * step;

        T p[25];
        int j1 = c >= 1 ? c - 1 : c;
        int j0 = c >= 2 ? c - 2 : j1;
        int j3 = c < COLS - 1 ? c + 1 : c;
        int j4 = c < COLS - 2 ? c + 2 : j3;
        for (int k = 0; k < 5; k++)
        {
            const T* rowk = row[k];
            p[k * 5] = rowk[j0]; p[k * 5 + 1] = rowk[j1];
            p[k * 5 + 2] = rowk[c]; p[k * 5 + 3] = rowk[j3];
            p[k * 5 + 4] = rowk[j4];
        }

        dst[r * step + c] = median25(p);
    }
}

void run_median_filter(
    dim3 grid,
    dim3 threads,
    hipStream_t cudaStream,
    const float *src,
    float *dst,
    const int ROWS,
    const int COLS,
    const int step,
    const int winSize) {
    kernel_median_filter<<<grid, threads, 0, cudaStream>>>(src, dst, ROWS, COLS, step, winSize);
}
